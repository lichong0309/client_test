#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <assert.h>
#include "ios/ops.h"
#include "utils/utils.h"

__global__ void assign_with_stride_dst(data_type *dst, const data_type *src, int n, int dst_blk_size, int src_blk_size) {
    CUDA_KERNEL_LOOP(i, n) {
        int blk_idx = i / dst_blk_size;
        int blk_offset = i % dst_blk_size;
        int src_offset = blk_idx * src_blk_size + blk_offset;
        int dst_offset = blk_idx * dst_blk_size + blk_offset;
        dst[dst_offset] = src[src_offset];
    }
}

__global__ void assign_with_stride_src(data_type *dst, const data_type *src, int n, int dst_blk_size, int src_blk_size) {
    CUDA_KERNEL_LOOP(i, n) {
        int blk_idx = i / src_blk_size;
        int blk_offset = i % src_blk_size;
        int src_offset = blk_idx * src_blk_size + blk_offset;
        int dst_offset = blk_idx * dst_blk_size + blk_offset;
        dst[dst_offset] = src[src_offset];
    }
}

__global__ void accumulate_sum_2(data_type *dst, const data_type *src1, const data_type *src2, int n) {
    CUDA_KERNEL_LOOP(i, n) {
        dst[i] = src1[i] + src2[i];
    }
}

__global__ void accumulate_sum_3(data_type *dst, const data_type *src1, const data_type *src2, const data_type *src3, int n) {
    CUDA_KERNEL_LOOP(i, n) {
        dst[i] = src1[i] + src2[i] + src3[i];
    }
}

__global__ void accumulate_sum_4(data_type *dst, const data_type *src1, const data_type *src2, const data_type *src3, const data_type *src4, int n) {
    CUDA_KERNEL_LOOP(i, n) {
        dst[i] = src1[i] + src2[i] + src3[i] + src4[i];
    }
}

__global__ void accumulate_sum_5(data_type *dst, const data_type *src1, const data_type *src2, const data_type *src3, const data_type *src4, const data_type *src5, int n) {
    CUDA_KERNEL_LOOP(i, n) {
        dst[i] = src1[i] + src2[i] + src3[i] + src4[i] + src5[i];
    }
}

